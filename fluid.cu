#include "hip/hip_runtime.h"
﻿#include "IO.h"
#include "Simulation.cuh"
#include "Renderer.cuh"



void Medium_Scale(int3 vol_d, int3 img_d, uint8_t* img, 
    float3 light, std::vector<OBJECT>& object_list, float3 cam, 
    int ACCURACY_STEPS, int FRAMES, int STEPS, float Dissolve_rate, 
    float Ambient_temp, float Fire_Max_Temp, bool Smoke_and_Fire,
    float time_step) {
    
    
    fluid_state state(vol_d);

    //VDB
    
    //VDB


    state.f_weight = 0.05;
    state.time_step = time_step;// 0.1;

    dim3 full_grid(vol_d.x / 8 + 1, vol_d.y / 8 + 1, vol_d.z / 8 + 1);
    dim3 full_block(8, 8, 8);

    bool DEBUG = true;
    for (int f = 0; f <= FRAMES; f++) {
        
        std::cout << "\rFrame " << f + 1 << "  -  ";

        render_fluid(
            img, img_d,
            state.density->readTarget(),
            state.temperature->readTarget(),
            vol_d, 1.0, light, cam, 0.0 * float(state.step),
            STEPS, Fire_Max_Temp, Smoke_and_Fire);

        save_image(img, img_d, "output/R" + pad_number(f + 1) + ".ppm");
        for (int st = 0; st < 1; st++) {
            simulate_fluid(state, object_list, ACCURACY_STEPS, DEBUG, f, Dissolve_rate, Ambient_temp);
            state.step++;
            //DEBUG = false;
        }

        GRID3D* arr = new GRID3D();
        GRID3D* arr_temp = new GRID3D();
        arr->set_pointer(state.density->readToGrid());
        arr_temp->set_pointer(state.temperature->readToGrid());
        /*
        */
        export_openvdb("frame."+std::to_string(f), vol_d, arr,arr_temp);
        arr->free();
        arr_temp->free();
    }

    delete[] img;

    printf("CUDA: %s\n", hipGetErrorString(hipGetLastError()));

    hipDeviceReset();
}
/*
void Huge_Scale(int3 vol_d, int3 img_d, uint8_t* img, float3 light, std::vector<OBJECT>& object_list, float3 cam, int ACCURACY_STEPS, int FRAMES, int STEPS) {
    fluid_state_huge state(vol_d);

    state.impulseLoc = make_float3(0.5 * float(vol_d.x),
        0.5 * float(vol_d.y) - 170.0,
        0.5 * float(vol_d.z));
    state.impulseTemp = 20.0;//4.0
    state.impulseDensity = 0.6;//0.35
    state.impulseRadius = 18.0;//18.0
    state.f_weight = 0.05;
    state.time_step = 0.1;

    dim3 full_grid(vol_d.x / 8 + 1, vol_d.y / 8 + 1, vol_d.z / 8 + 1);
    dim3 full_block(8, 8, 8);


    for (int f = 0; f <= FRAMES; f++) {

        std::cout << "\rFrame " << f + 1 << "  -  ";

        if (_kbhit()) {
            std::cout << "Stopping simulation\n";
            break;
        }

        render_fluid(
            img, img_d,
            state.density->readTarget(),
            state.temperature->readTarget(),
            vol_d, 1.0, light, cam, 0.0 * float(state.step),
            STEPS);

        save_image(img, img_d, "output/R" + pad_number(f + 1) + ".ppm");
        for (int st = 0; st < 1; st++) {
            simulate_fluid(state, object_list, ACCURACY_STEPS);
            state.step++;
        }
    }

    delete[] img;

    printf("CUDA: %s\n", hipGetErrorString(hipGetLastError()));

    hipDeviceReset();
}
*/
int main(int argc, char* args[])
{
    openvdb::initialize();
    srand(0);
    //simulation settings
    int3 DOMAIN_RESOLUTION = make_int3(256,600,256);
    int ACCURACY_STEPS = 8; //8
    std::vector<OBJECT> object_list;

    float Smoke_Dissolve = 0.995f; //0.995f
    float Ambient_Temperature = 0.0f; //0.0f
    float speed = 1.0; //1.0

    


    //rendering settings
    int FRAMES = 500;
    float Fire_Max_Temperature = 50.0f;
    float Image_Resolution[2] = { 640, 640 };
    int STEPS = 100; //512 Rendering Samples
    float ZOOM = 0.45; //1.8
    bool Smoke_And_Fire = true;



    float time_step = 0.1; //0.1
    time_step = speed * 0.1; //chyba dobre


    const int3 vol_d = make_int3(DOMAIN_RESOLUTION.x, DOMAIN_RESOLUTION.y, DOMAIN_RESOLUTION.z); //Domain resolution
    const int3 img_d = make_int3(Image_Resolution[0], Image_Resolution[1], 0);




    



    /////////VDB
    export_vdb("sphere",vol_d);



    clock_t startTime = clock();
    GRID3D sphere = load_vdb("sphere",vol_d);
    std::cout << "Loaded in : "<< double(clock() - startTime) / (double)CLOCKS_PER_SEC<< "s" << std::endl;

    if (false) {
        OBJECT SPHERE("vdb", 18.0f, 50, 0.9, 5, 0.9, make_float3(vol_d.x * 0.25, 10.0, 200.0));
        SPHERE.load_density_grid(sphere, 3.0);
        object_list.push_back(SPHERE);
    }
    else {
        OBJECT SPHERE("vdbsingle", 18.0f, 50, 0.9, 5, 0.9, make_float3(vol_d.x * 0.25, 10.0, 200.0));
        SPHERE.load_density_grid(sphere, 6.0);
        object_list.push_back(SPHERE);
    }
    
    
    
    //renderImage("sphere", 2);
    //exit(1);
    ////////////////

    //adding emmiters
    //object_list.push_back(OBJECT("emmiter", 18.0f, 50, 0.9, 5 ,0.9, make_float3(vol_d.x * 0.25, 10.0, 200.0)));
    //object_list.push_back(OBJECT("emmiter", 18.0f, 50, 0.6, 5, 0.9, make_float3(vol_d.x * 0.5, 10.0, 200.0)));
    //object_list.push_back(OBJECT("emmiter", 18.0f, 50, 0.3, 5, 0.9, make_float3(vol_d.x * 0.75, 10.0, 200.0)));
    //object_list.push_back(OBJECT("smoke", 10, 50, 0.9, 50, 1.0, make_float3(vol_d.x * 0.5, 10.0, 200.0)));




    float3 cam;
    cam.x = static_cast<float>(vol_d.x) * 0.5;
    cam.y = static_cast<float>(vol_d.y) * 0.5;
    cam.z = static_cast<float>(vol_d.z) * -0.4 * (1.0 / ZOOM);//0.0   minus do ty�u, plus do przodu
    float3 light;
    //X - lewo prawo
    //Y - g�ra d�
    //Z - prz�d ty�
    light.x = 5.0;//0.1
    light.y = 1.0;//1.0
    light.z = -0.5;//-0.5

    uint8_t* img = new uint8_t[3 * img_d.x * img_d.y];

    std::cout << "Clearing previous frames\n";
    std::system("erase_imgs.sh");
    std::system("rm ./output/cache/*");

    if (DOMAIN_RESOLUTION.x*DOMAIN_RESOLUTION.y*DOMAIN_RESOLUTION.z <= 100000000)
        Medium_Scale(vol_d, img_d, img, light, object_list, cam, ACCURACY_STEPS, FRAMES, STEPS, Smoke_Dissolve, Ambient_Temperature, Fire_Max_Temperature, Smoke_And_Fire, time_step);
    else {
        std::cout << "Domain resolution over 450^3 not supported yet" << std::endl;
        //Huge_Scale(vol_d, img_d, img, light, object_list, cam, ACCURACY_STEPS, FRAMES, STEPS);
    }

    for (int i = 0; i < object_list.size(); i++) { //czyszczenie pamięci GPU
        object_list[i].hipFree();
    }
    
    std::cout << "Rendering animation video..." << std::endl;
    std::system("make_video.sh");
    //std::system("pause");

    return 0;
}
