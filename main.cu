#include "hip/hip_runtime.h"
﻿#define GUI


//#include "Solver.cuh"
#include "Window.h"

extern Solver solver;



/////////////////////////////////
//section experimental
void EnableP2Psharing(unsigned int devices_count = 1) {
    std::cout << "Enabling P2P sharing..." << std::endl;
    for (unsigned int i = 0; i < devices_count; i++) {
        for (unsigned int j = 0; j < devices_count; j++) {
            int is_able = NULL;
            hipSetDevice(i);
            hipDeviceCanAccessPeer(&is_able, i, j);
            if (is_able) {
                checkCudaErrors(hipDeviceEnablePeerAccess(j, 0));
                std::cout << "Enabled P2P sharing for: " << i << std::endl;
            }
        }
    }
}

/////////////////////////////////



int main(int argc, char* argv[]) {
    //srand(1);
    int devicesCount;
    hipGetDeviceCount(&devicesCount);
    std::cout << "Found " << devicesCount << " devices:" << std::endl;
    std::cout << "----------------------------------------" << std::endl;
    for (int deviceIndex = 0; deviceIndex < devicesCount; ++deviceIndex)
    {
        hipDeviceProp_t deviceProperties;
        hipGetDeviceProperties(&deviceProperties, deviceIndex);
        std::cout << deviceProperties.name << "   ->  " << deviceProperties.totalGlobalMem << std::endl;
    }
    int Best_Device_Index = 0;
    long long Memory = 0;
    for (int deviceIndex = 0; deviceIndex < devicesCount; deviceIndex++) {
        hipDeviceProp_t deviceProperties;
        hipGetDeviceProperties(&deviceProperties, deviceIndex);
        if (deviceProperties.totalGlobalMem > Memory){
            Memory = deviceProperties.totalGlobalMem;
            Best_Device_Index = deviceIndex;
        }
    }
    std::cout << "----------------------------------------" << std::endl;
    hipSetDevice(Best_Device_Index);
    std::cout << "Choosing device: " << Best_Device_Index << std::endl;

    if (false) {
        hipDeviceProp_t deviceProperties;
        hipGetDeviceProperties(&deviceProperties, Best_Device_Index);
        hipDeviceSetLimit(cudaLimitPersistingL2CacheSize, deviceProperties.persistingL2CacheMaxSize); /* Set aside max possible size of L2 cache for persisting accesses */
        std::cout << "Setting L2 max cache: " << deviceProperties.persistingL2CacheMaxSize << std::endl;
    }

    EnableP2Psharing(devicesCount);

#ifdef EXPERIMENTAL
    if (argc <= 1) {
        if (false) {
            std::cout << "Using All (" << devicesCount << ") devices" << std::endl;
            solver.Initialize(devicesCount);
        }
        else {
            std::cout << "Using (" << 1 << ") device" << std::endl;
            solver.Initialize(1);
        }
    }
    else {
        std::cout << "Using " << std::stoi(argv[1]) << " devices" << std::endl;
        solver.Initialize(std::stoi(argv[1]));
    }
#ifdef OBJECTS_EXPERIMENTAL
    std::cout << "Generating example scene" << std::endl;
    solver.ExampleScene(true);
#else
    solver.ExportVDBScene();
#endif
    //solver.ExampleScene(true);
    float Window_Resolution[2] = { 1600, 800 };
    float Image_Resolution[2] = { 700, 900 };
    std::cout << "Setting image resolution" << std::endl;
    solver.setImageResolution(Image_Resolution[0], Image_Resolution[1]);

    solver.Initialize_Simulation();
    Window(Window_Resolution);
    solver.Clear_Simulation_Data();
    //std::cout << "Rendering animation video..." << std::endl;
    //std::system("make_video.sh");
#else
#ifdef GUI

    std::cout << "Hello" << std::endl;

    float Image_Resolution[2] = { 640, 640 };
    const int3 img_d = make_int3(Image_Resolution[0], Image_Resolution[1], 0);

    uint8_t* img = new uint8_t[3 * img_d.x * img_d.y];

    Window(Image_Resolution);
#else
    initialize();
#endif
#endif
    return 0;
}