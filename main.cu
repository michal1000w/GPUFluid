#include "hip/hip_runtime.h"
﻿#define GUI


//#include "Solver.cuh"
#include "Window.h"

extern Solver solver;



int main(int argc, char* args[]) {
    int devicesCount;
    hipGetDeviceCount(&devicesCount);
    for (int deviceIndex = 0; deviceIndex < devicesCount; ++deviceIndex)
    {
        hipDeviceProp_t deviceProperties;
        hipGetDeviceProperties(&deviceProperties, deviceIndex);
        std::cout << deviceProperties.name << "   ->  " << deviceProperties.totalGlobalMem << std::endl;
    }
    int Best_Device_Index = 0;
    int Memory = 0;
    for (int deviceIndex = 0; deviceIndex < devicesCount; ++deviceIndex) {
        hipDeviceProp_t deviceProperties;
        hipGetDeviceProperties(&deviceProperties, deviceIndex);
        if (deviceProperties.totalGlobalMem > Memory){
            Memory = deviceProperties.totalGlobalMem;
            Best_Device_Index = deviceIndex;
        }
    }
    hipSetDevice(Best_Device_Index);
#ifdef EXPERIMENTAL
    solver.Initialize();
    //solver.ClearCache();
#ifdef OBJECTS_EXPERIMENTAL
    solver.ExampleScene();
#else
    solver.ExportVDBScene();
#endif
    //solver.ExampleScene();
    float Window_Resolution[2] = { 1600, 640 };
    float Image_Resolution[2] = { 640, 640 };
    solver.setImageResolution(Image_Resolution[0], Image_Resolution[1]);

    solver.Initialize_Simulation();
    Window(Window_Resolution);
    solver.Clear_Simulation_Data();
    //std::cout << "Rendering animation video..." << std::endl;
    //std::system("make_video.sh");
#else
#ifdef GUI

    std::cout << "Hello" << std::endl;

    float Image_Resolution[2] = { 640, 640 };
    const int3 img_d = make_int3(Image_Resolution[0], Image_Resolution[1], 0);

    uint8_t* img = new uint8_t[3 * img_d.x * img_d.y];

    Window(Image_Resolution);
#else
    initialize();
#endif
#endif
    return 0;
}