#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <string>
#include <fstream>
#include <iostream>
#include <sstream>
#include <iomanip>
#include <vector>
//#include <hip/hip_fp16.h>
#include "cutil_math.h"



#define _USE_MATH_DEFINES
#include <cmath>
#include <openvdb/openvdb.h>
#include <openvdb/tools/LevelSetSphere.h>
#include <openvdb/tools/SignedFloodFill.h>
#include <openvdb/io/Stream.h>
#include <openvdb/io/Compression.h>
#include <openvdb/tree/ValueAccessor.h>
////////////////////////////////////////////
#include "third_party/openvdb/nanovdb/nanovdb/NanoVDB.h"
#include <windows.h>
#include <ppl.h>
#include <thread>
//#include "OpenVDB-old/tinyvdbio.h"
#include <nanovdb/util/GridBuilder.h>
#include <nanovdb/util/IO.h>
#include <nanovdb/util/CudaDeviceBuffer.h>
#include <nanovdb/util/GridBuilder.h>
////////////////////////////////////////////







#include "Fluid_Kernels.cuh"
#include "Unified_Buffer.cpp"

//#define EXPERIMENTAL
//#include <driver_functions.h>
//#include <hip/driver_types.h>



struct fluid_state_huge {

    float3 impulseLoc;
    float impulseTemp;
    float impulseDensity;
    float impulseRadius;
    float f_weight;
    float cell_size;
    float time_step;
    int3 dim;
    int64_t nelems;
    int step;
    UnifiedBuffer<float3>* velocity;
    UnifiedBuffer<float>* density;
    UnifiedBuffer<float>* temperature;
    UnifiedBuffer<float>* pressure;
    float* diverge;

    fluid_state_huge(int3 dims) {
        step = 0;
        dim = dims;
        nelems = dims.x * dims.y * dims.z;
        velocity = new UnifiedBuffer<float3>((int)nelems);
        density = new UnifiedBuffer<float>((int)nelems);
        temperature = new UnifiedBuffer<float>((int)nelems);
        pressure = new UnifiedBuffer<float>((int)nelems);
        

        hipMalloc((void**)&diverge, sizeof(float) * nelems);
        //hipDeviceSynchronize();
    }

    ~fluid_state_huge() {
        delete velocity;
        delete density;
        delete temperature;
        delete pressure;
        hipFree(diverge);
    }
};
